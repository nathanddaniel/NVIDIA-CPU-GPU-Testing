#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h" //CUDA API functions
#include <string.h> 
#include <stdio.h> //used for printing stuff for like name of CUDA able GPUs


int main() {

    int nd; //number of avialble CUDA GPUs

    //the function hipGetDeviceCount() returns the number of CUDA capable GPUs
    hipGetDeviceCount(&nd);

    //printing out the number of CUDA capable GPUs available
    printf("Number of CUDA capable GPUs that are available is: %d\n", nd);

    //now we're going to loop through each CUDA GPU 
    for (int i = 0; i < nd; i++){

        /* we're creating a variable dp of type hipDeviceProp_t which
        is a struct that contains the properties of the CUDA GPU
        */
        hipDeviceProp_t dp;

        //filling in the structure with the properties of the GPU
        hipGetDeviceProperties(&dp, i);

        //printing out the GPU info
        printf("Device %d --> Max Threads per SM: %d, Warp Size: %d\n", i, dp.maxThreadsPerMultiProcessor, dp.warpSize);

        //printing out the clock info
        printf("Clock Rate is: %d kHz\n", dp.clockRate);

        //printing out the number of SM's
        printf("Number of Streaming Multiprocessors is: %d\n", dp.multiProcessorCount);

        //printing out the number of cores
        //should I do CUDA Core per SM count for my GPU model?

        //printing the amount of Global Memory
        printf("Amount of Global Memory is: %d bytes \n", dp.totalGlobalMem);

        
        //printing the amount of Constant Memory
        printf("Amount of Constant Memory is: %d bytes \n", dp.totalConstMem);

        //printing the GPU Model Name
        printf("GPU Name: %s\n", dp.name);

        //printing the shared memory for each block
        printf("Shared Memory Per Block: %d \n", dp.sharedMemPerBlock);

        //printing the number of registers per block
        printf("Number of registers/block: %d \n", dp.regsPerBlock);

        //printing the maximum number of threads per block
        printf("Max number of threads per block: %d \n", dp.maxThreadsPerBlock);
    }

    return 0;
}
