#include <stdio.h>
#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <time.h>

// CPU Matrix Multiplication
__host__ void matrixMulCPU(float* P, float* M, float* N, int Nsize) {
    for (int i = 0; i < Nsize; i++) {
        for (int j = 0; j < Nsize; j++) {
            float sum = 0;
            for (int k = 0; k < Nsize; k++) {
                sum += M[i * Nsize + k] * N[k * Nsize + j];
            }
            P[i * Nsize + j] = sum;
        }
    }
}

// GPU Matrix Multiplication Kernel
__global__ void MatrixMulKernel(float* M, float* N, float* P, int Width) {
    int Row = blockIdx.y * blockDim.y + threadIdx.y;
    int Col = blockIdx.x * blockDim.x + threadIdx.x;

    if (Row < Width && Col < Width) {
        float Pvalue = 0;
        for (int k = 0; k < Width; ++k) {
            Pvalue += M[Row * Width + k] * N[k * Width + Col];
        }
        P[Row * Width + Col] = Pvalue;
    }
}

// Function to measure Host → Device transfer time
float CPUtoGPUTime(float* d_ptr, float* h_ptr, int size) {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    hipMemcpy(d_ptr, h_ptr, size, hipMemcpyHostToDevice);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);
    return milliseconds;
}

// Function to measure Device → Host transfer time
float GPUtoCPUTime(float* h_ptr, float* d_ptr, int size) {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    hipMemcpy(h_ptr, d_ptr, size, hipMemcpyDeviceToHost);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);
    return milliseconds;
}

// Function to measure GPU matrix multiplication execution time
float GPUExecutionTime(float* d_M, float* d_N, float* d_P, int Nsize) {
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((Nsize + threadsPerBlock.x - 1) / threadsPerBlock.x, 
                   (Nsize + threadsPerBlock.y - 1) / threadsPerBlock.y);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    MatrixMulKernel<<<numBlocks, threadsPerBlock>>>(d_M, d_N, d_P, Nsize);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);
    return milliseconds;
}

int main() {
    int matrixSizes[] = {256, 512, 1024, 2048, 4096};
    int numOfSizes = sizeof(matrixSizes) / sizeof(matrixSizes[0]);
    int floatSize = sizeof(float);

    for (int i = 0; i < numOfSizes; i++) {
        int N = matrixSizes[i];
        int bytes = N * N * floatSize;

        // Allocate memory on CPU
        float *h_M = (float*)malloc(bytes);
        float *h_N = (float*)malloc(bytes);
        float *h_P_cpu = (float*)malloc(bytes);
        float *h_P_gpu = (float*)malloc(bytes);

        // Initialize matrices with random values
        for (int j = 0; j < N * N; j++) {
            h_M[j] = (float)(rand() % 5);
            h_N[j] = (float)(rand() % 5);
        }

        // Allocate memory on GPU
        float *d_M, *d_N, *d_P;
        hipMalloc(&d_M, bytes);
        hipMalloc(&d_N, bytes);
        hipMalloc(&d_P, bytes);

        // Measure CPU execution time
        clock_t start_cpu = clock();
        matrixMulCPU(h_P_cpu, h_M, h_N, N);
        clock_t end_cpu = clock();
        float cpu_time = 1000.0 * (end_cpu - start_cpu) / CLOCKS_PER_SEC;

        // Measure Host → Device transfer time
        float h2d_time_M = CPUtoGPUTime(d_M, h_M, bytes);
        float h2d_time_N = CPUtoGPUTime(d_N, h_N, bytes);
        float total_h2d_time = h2d_time_M + h2d_time_N;

        // Measure GPU execution time
        float gpu_time = GPUExecutionTime(d_M, d_N, d_P, N);

        // Measure Device → Host transfer time
        float d2h_time_P = GPUtoCPUTime(h_P_gpu, d_P, bytes);

        // Total GPU time including memory transfers
        float total_gpu_time = total_h2d_time + gpu_time + d2h_time_P;

        // Print results
        printf("Matrix Size: %d x %d\n", N, N);
        printf("CPU Execution Time: %.3f ms\n", cpu_time);
        printf("GPU Execution Time (Without Transfers): %.3f ms\n", gpu_time);
        printf("GPU Execution Time (With Transfers): %.3f ms\n\n", total_gpu_time);

        // Free memory
        free(h_M);
        free(h_N);
        free(h_P_cpu);
        free(h_P_gpu);
        hipFree(d_M);
        hipFree(d_N);
        hipFree(d_P);
    }

    return 0;
}