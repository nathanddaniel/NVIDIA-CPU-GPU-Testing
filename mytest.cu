#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <cmath>
#include <vector>

#define MAX_BLOCK_WIDTH 32  // Maximum block width to test
#define EPSILON 1e-5        // Tolerance for result comparison

// **Single-Threaded GPU Kernel**
__global__ void kernelMatMul_SingleThread(float* P, float* M, float* N, int size) {
    if (threadIdx.x == 0 && threadIdx.y == 0 && blockIdx.x == 0 && blockIdx.y == 0) {
        for (int row = 0; row < size; row++) {
            for (int col = 0; col < size; col++) {
                float sum = 0.0;
                for (int i = 0; i < size; i++) {
                    sum += M[row * size + i] * N[i * size + col];
                }
                P[row * size + col] = sum;
            }
        }
    }
}

// **Multi-Threaded GPU Kernel**
__global__ void kernelMatMul_MultiThread(float* P, float* M, float* N, int size) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < size && col < size) {
        float sum = 0.0;
        for (int i = 0; i < size; i++) {
            sum += M[row * size + i] * N[i * size + col];
        }
        P[row * size + col] = sum;
    }
}

// **CPU Matrix Multiplication**
void cpuMatMul(float* P, float* M, float* N, int size) {
    for (int row = 0; row < size; row++) {
        for (int col = 0; col < size; col++) {
            float sum = 0.0;
            for (int i = 0; i < size; i++) {
                sum += M[row * size + i] * N[i * size + col];
            }
            P[row * size + col] = sum;
        }
    }
}

int main() {
    std::vector<int> sizes = {256, 512, 1024, 2048, 4096};
    std::vector<int> blockWidths = {2, 4, 8, 16, 32};

    printf("\n========= Data Transfer Time (Host to Device) =========\n");
    for (int size : sizes) {
        int matrixSize = size * size;
        float *hostM, *hostN, *deviceM, *deviceN;

        // Allocate memory
        hostM = (float*)malloc(matrixSize * sizeof(float));
        hostN = (float*)malloc(matrixSize * sizeof(float));
        hipMalloc(&deviceM, matrixSize * sizeof(float));
        hipMalloc(&deviceN, matrixSize * sizeof(float));

        // Initialize matrices
        for (int i = 0; i < matrixSize; i++) {
            hostM[i] = static_cast<float>(rand()) / RAND_MAX;
            hostN[i] = static_cast<float>(rand()) / RAND_MAX;
        }

        // Timing events
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        // **Host to Device Transfer**
        float timeH2D;
        hipEventRecord(start, 0);
        hipMemcpy(deviceM, hostM, matrixSize * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(deviceN, hostN, matrixSize * sizeof(float), hipMemcpyHostToDevice);
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&timeH2D, start, stop);
        printf("Matrix Size: %d x %d | Host to Device: %.3f ms\n", size, size, timeH2D);

        // **Device to Host Transfer**
        float timeD2H;
        hipEventRecord(start, 0);
        hipMemcpy(hostM, deviceM, matrixSize * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(hostN, deviceN, matrixSize * sizeof(float), hipMemcpyDeviceToHost);
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&timeD2H, start, stop);
        printf("Matrix Size: %d x %d | Device to Host: %.3f ms\n", size, size, timeD2H);

        // Cleanup
        free(hostM);
        free(hostN);
        hipFree(deviceM);
        hipFree(deviceN);
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    printf("\n========= Single-Threaded GPU vs. CPU Computation =========\n");
    for (int size : {256, 512, 1024}) {
        int matrixSize = size * size;

        // **Allocate Host Memory for Input & Output Matrices**
        float *hostM, *hostN, *cpuHostP, *kernelHostP;
        hostM = (float*)malloc(matrixSize * sizeof(float));
        hostN = (float*)malloc(matrixSize * sizeof(float));
        cpuHostP = (float*)malloc(matrixSize * sizeof(float));  // CPU output
        kernelHostP = (float*)malloc(matrixSize * sizeof(float)); // GPU output

        // **Allocate Device Memory**
        float *deviceM, *deviceN, *deviceP;
        hipMalloc(&deviceM, matrixSize * sizeof(float));
        hipMalloc(&deviceN, matrixSize * sizeof(float));
        hipMalloc(&deviceP, matrixSize * sizeof(float));

        // **Initialize Input Matrices**
        for (int i = 0; i < matrixSize; i++) {
            hostM[i] = static_cast<float>(rand()) / RAND_MAX;
            hostN[i] = static_cast<float>(rand()) / RAND_MAX;
        }

        // **Copy Matrices to Device**
        hipMemcpy(deviceM, hostM, matrixSize * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(deviceN, hostN, matrixSize * sizeof(float), hipMemcpyHostToDevice);

        // **Single-Threaded GPU Execution**
        dim3 gridDim(1, 1);
        dim3 blockDim(1, 1);

        float gpuTime;
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        
        hipEventRecord(start, 0);
        kernelMatMul_SingleThread<<<gridDim, blockDim>>>(deviceP, deviceM, deviceN, size);
        hipDeviceSynchronize();
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&gpuTime, start, stop);
        
        printf("Matrix Size: %d x %d | Single-Threaded GPU: %.3f ms\n", size, size, gpuTime);

        // **Copy GPU Result to Host**
        hipMemcpy(kernelHostP, deviceP, matrixSize * sizeof(float), hipMemcpyDeviceToHost);

        // **CPU Execution Timing**
        float cpuTime;
        hipEventRecord(start, 0);
        cpuMatMul(cpuHostP, hostM, hostN, size);
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&cpuTime, start, stop);

        printf("Matrix Size: %d x %d | CPU Computation: %.3f ms\n", size, size, cpuTime);

        // **Compare CPU and GPU Results**
        bool isCorrect = true;
        for (int i = 0; i < matrixSize; i++) {
            if (fabs(kernelHostP[i] - cpuHostP[i]) > EPSILON) {
                printf("Mismatch at index %d: CPU = %.6f, GPU = %.6f\n", i, cpuHostP[i], kernelHostP[i]);
                isCorrect = false;
                break;  // Stop checking after first mismatch (optional)
            }
        }

        // **Print Final Verification Result**
        if (isCorrect) {
            printf("Test PASSED\n");
        } else {
            printf("Test FAILED\n");
        }

        // **Free Memory**
        free(hostM);
        free(hostN);
        free(cpuHostP);
        free(kernelHostP);
        hipFree(deviceM);
        hipFree(deviceN);
        hipFree(deviceP);
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    printf("\n========= Multi-Threaded GPU Computation =========\n");
    for (int size : sizes) {
        int matrixSize = size * size;
        float *deviceP;
        hipMalloc(&deviceP, matrixSize * sizeof(float));

        for (int blockWidth : blockWidths) {
            dim3 gridDim((size + blockWidth - 1) / blockWidth, (size + blockWidth - 1) / blockWidth);
            dim3 blockDim(blockWidth, blockWidth);

            // **Multi-Threaded GPU Execution**
            float multiThreadTime;
            hipEvent_t start, stop;
            hipEventCreate(&start);
            hipEventCreate(&stop);
            hipEventRecord(start, 0);
            kernelMatMul_MultiThread<<<gridDim, blockDim>>>(deviceP, deviceP, deviceP, size);
            hipDeviceSynchronize();
            hipEventRecord(stop, 0);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&multiThreadTime, start, stop);
            printf("Matrix Size: %d x %d | Block Width: %d | Multi-Threaded GPU: %.3f ms\n",
                   size, size, blockWidth, multiThreadTime);
            hipEventDestroy(start);
            hipEventDestroy(stop);
        }

        hipFree(deviceP);
    }

    return 0;
}